#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <map>
#include "parse.h"
#include "sequential.h"
#include "util.cuh"
#include "kernels.cuh"

#include "main.cuh"

std::map<int, double> compute_bc(std::vector<std::pair<int, int> > edges, int k, bool approx,	int device ){
	int max_threads_per_block, number_of_SMs;
	choose_device(max_threads_per_block,number_of_SMs, device);
	graph g = parse_edgelist(edges);

	std::cout << "Number of nodes: " << g.n << std::endl;
	std::cout << "Number of edges: " << g.m << std::endl;

	//If we're approximating, choose source vertices at random
	std::set<int> source_vertices;
	if(approx)
	{
		if(k > g.n || k < 1)
		{
			k = g.n;
		}

		while(source_vertices.size() < k)
		{
			int temp_source = rand() % g.n;
			source_vertices.insert(temp_source);
		}
	}

	hipEvent_t start,end;
	float CPU_time;
	std::vector<float> bc;
	if(false) //Only run CPU code if verifying
	{
		start_clock(start,end);
		bc = bc_cpu(g,source_vertices);
		CPU_time = end_clock(start,end);
	}

	float GPU_time;
	std::vector<double> bc_g;
	start_clock(start,end);
	bc_g = bc_gpu(g,max_threads_per_block,number_of_SMs,approx, k,source_vertices);
	GPU_time = end_clock(start,end);

	if(false)
	{
		//verify(g,bc,bc_g);
	}
	if(false)
	{
		//g.print_BC_scores(bc_g,"summa.txt");
	}

	std::cout << std::setprecision(9);
	if(false)
	{
		std::cout << "Time for CPU Algorithm: " << CPU_time << " s" << std::endl;
	}
	std::cout << "Time for GPU Algorithm: " << GPU_time << " s" << std::endl;

	std::map<int, double> bc_scores = g.get_BC_scores(bc_g);
	
	delete[] g.R;
	delete[] g.C;
	delete[] g.F;
	return bc_scores;
}