#include "hip/hip_runtime.h"
#include "util.cuh"

//Note: Times are returned in seconds
void start_clock(hipEvent_t &start, hipEvent_t &end)
{
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&end));
	checkCudaErrors(hipEventRecord(start,0));
}

float end_clock(hipEvent_t &start, hipEvent_t &end)
{
	float time;
	checkCudaErrors(hipEventRecord(end,0));
	checkCudaErrors(hipEventSynchronize(end));
	checkCudaErrors(hipEventElapsedTime(&time,start,end));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(end));

	return time/(float)1000;
}


void choose_device(int &max_threads_per_block, int &number_of_SMs, int device)
{
	int count;
	checkCudaErrors(hipGetDeviceCount(&count));
	hipDeviceProp_t prop;

	if(device == -1)
	{
		int maxcc=0, bestdev=0;
		for(int i=0; i<count; i++)
		{
			checkCudaErrors(hipGetDeviceProperties(&prop,i));
			if((prop.major + 0.1*prop.minor) > maxcc)
			{
				maxcc = prop.major + 0.1*prop.minor;
				bestdev = i;
			}	
		}

		checkCudaErrors(hipSetDevice(bestdev));
		checkCudaErrors(hipGetDeviceProperties(&prop,bestdev));
	}
	else if((device < -1) || (device >= count))
	{
		std::cerr << "Invalid device argument. Valid devices on this machine range from 0 through " << count-1 << "." << std::endl;
		exit(-1);
	}
	else
	{
		checkCudaErrors(hipSetDevice(device));
		checkCudaErrors(hipGetDeviceProperties(&prop,device));
	}

	std::cout << "Chosen Device: " << prop.name << std::endl;
	std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
	std::cout << "Number of Streaming Multiprocessors: " << prop.multiProcessorCount << std::endl;
	std::cout << "Size of Global Memory: " << prop.totalGlobalMem/(float)(1024*1024*1024) << " GB" << std::endl << std::endl;

	max_threads_per_block = prop.maxThreadsPerBlock;
	number_of_SMs = prop.multiProcessorCount;
}

void verify(graph g, const std::vector<float> bc_cpu, const std::vector<float> bc_gpu)
{
	double error = 0;
	double max_error = 0;
	for(int i=0; i<g.n; i++)
	{
		double current_error = abs(bc_cpu[i] - bc_gpu[i]);
		error += current_error*current_error;
		if(current_error > max_error)
		{
			max_error = current_error;
		}
	}
	error = error/(float)g.n;
	error = sqrt(error);
	std::cout << "RMS Error: " << error << std::endl;
	std::cout << "Maximum error: " << max_error << std::endl;
}
